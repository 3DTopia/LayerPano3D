#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

std::tuple<torch::Tensor, torch::Tensor> distCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  auto int_opts = points.options().dtype(torch::kInt32);
  torch::Tensor nearestIndices = torch::empty({P, 3}, int_opts).set_requires_grad(false);
  torch::Tensor means = torch::full({P}, 0.0, float_opts);
  
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>(), nearestIndices.contiguous().data<int32_t>());

  return std::make_tuple(means, nearestIndices);
}